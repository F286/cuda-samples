#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <assert.h>
#include <memory>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include "ML_Array.h"

struct ML_DenseConnection;

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const float *A, const float *B, float *C,
                          int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements) {
    C[i] = A[i] + B[i] + 0.0f;
  }
}

template <class ConnectionType>
__global__ void vectorMultiply(const ML_DeviceArray<float> A, const ML_DeviceArray<float> B, ML_DeviceArray<float> C) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < Int2::Size(C.numElements)) 
    {
        float* bufferRootB = &B.deviceBuffer[i * A.numElements.x];

        float total = 0.0f;
        for (int elementIndex = 0; elementIndex < A.numElements.x; elementIndex++)
        {
            total += ConnectionType::Run(A.deviceBuffer[elementIndex], bufferRootB[elementIndex]);
        }

        C.deviceBuffer[i] = total;
    }
}

// Possible to pass in method to execute on cuda vectors with template using global method?
void Multiply(ML_Array<float>& arrayA, ML_Array<float>& arrayB, ML_Array<float>& arrayOut)
{
    assert(arrayB.NumElements().x == arrayA.NumElements().x);
    assert(arrayB.NumElements().y == arrayOut.NumElements().x);

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    Int2 numElements = arrayOut.deviceArray.numElements;
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements.Size() + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
        threadsPerBlock);
    vectorMultiply<ML_DenseConnection> << <blocksPerGrid, threadsPerBlock >> > (arrayA.deviceArray, arrayB.deviceArray, arrayOut.deviceArray);
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

struct ML_DenseConnection
{
    ML_DenseConnection(ML_Array<float>& previous, ML_Array<float>& next)
        : previous(previous)
        , next(next)
        , connection(ConnectionMatrixSize(previous, next))
    {
    }

    void Run()
    {
        previous.HostToDevice();
        connection.HostToDevice();

        // Launch the Vector Multiply CUDA Kernel
        Multiply(previous, connection, next);

        next.DeviceToHost();
    }

    static Int2 ConnectionMatrixSize(const ML_Array<float>& previous, const ML_Array<float>& next)
    {
        assert(previous.NumElements().y == 1);
        assert(next.NumElements().y == 1);
        return Int2{ previous.NumElements().x, next.NumElements().x };
    }

    __host__ __device__ static float Run(const float& previous, const float& connection)
    {
        return previous * connection;
    }

    float& operator[] (int index)
    {
        return connection[index];
    }
    float& operator[] (Int2 position)
    {
        return connection[position];
    }

    ML_Array<float>& previous;
    ML_Array<float>& next;

    ML_Array<float> connection;
};

struct ML_ValueDecorator
{
};
struct ML_ConnectionDecorator
{
};

struct ML_DenseArrayDerivative : public ML_ValueDecorator
{
    // TODO (fd) : ML_Array should be an object that supports weights and biases. Struct packed together for memory access efficiency.

    ML_Array<float>& original;
    ML_Array<float> derivative;
};

struct ML_DenseConnectionDerivative : public ML_ConnectionDecorator
{

};

void Run()
{
    ML_Array<float> array1{ Int2{ 3, 1 } };
    array1[Int2{ 0, 0 }] = 10;
    array1[Int2{ 1, 0 }] = 100;
    array1[Int2{ 2, 0 }] = 1000;

    ML_Array<float> array2{ Int2{ 4, 1 } };

	ML_DenseConnection connection1{ array1, array2 };

    connection1[Int2{ 0, 0 }] = 1;
    connection1[Int2{ 1, 0 }] = 0;
    connection1[Int2{ 2, 0 }] = 0;

    connection1[Int2{ 0, 1 }] = 0;
    connection1[Int2{ 1, 1 }] = 1;
    connection1[Int2{ 2, 1 }] = 0;

    connection1[Int2{ 0, 2 }] = 0;
    connection1[Int2{ 1, 2 }] = 0;
    connection1[Int2{ 2, 2 }] = 1;

    connection1[Int2{ 0, 3 }] = 1;
    connection1[Int2{ 1, 3 }] = 1;
    connection1[Int2{ 2, 3 }] = 1;

    connection1.Run();

    // Verify that the result vector is correct
    assert(array2[0] == 10);
    assert(array2[1] == 100);
    assert(array2[2] == 1000);
    assert(array2[3] == 1110);

    printf("Test PASSED\n");
}

/**
 * Host main routine
 */
int main(void) {

    Run();

    printf("Done\n");
    return 0;
}
