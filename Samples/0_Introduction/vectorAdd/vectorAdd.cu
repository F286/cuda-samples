#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include "ML_Array.h"

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const float *A, const float *B, float *C,
                          int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements) {
    C[i] = A[i] + B[i] + 0.0f;
  }
}

__global__ void vectorMultiply(const ML_DeviceArray A, const ML_DeviceArray B, ML_DeviceArray C) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < C.numElements.x * C.numElements.y) {
        C.deviceBuffer[i] = A.deviceBuffer[i] * B.deviceBuffer[i] + 0.0f;
    }
}

void Multiply(ML_Array& arrayA, ML_Array& arrayB, ML_Array& arrayOut)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    Int2 numElements = arrayA.deviceArray.numElements;
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements.Size() + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
        threadsPerBlock);
    vectorMultiply << <blocksPerGrid, threadsPerBlock >> > (arrayA.deviceArray, arrayB.deviceArray, arrayOut.deviceArray);
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void Run()
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    Int2 numElements{ 2, 1 };
    size_t size = numElements.Size() * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements.Size());

    ML_Array arrayA{ Int2{3, 1} };
    //arrayA.InitializeToRandomValues();
    arrayA[Int2{ 0, 0 }] = 2;
    arrayA[Int2{ 1, 0 }] = 3;
    arrayA[Int2{ 2, 0 }] = 4;

    ML_Array arrayB{ Int2{3, 1} };
    //ML_Array arrayB{ Int2{3, 3} };
    //arrayB.InitializeToRandomValues();
    arrayB[Int2{ 0, 0 }] = 1;
    arrayB[Int2{ 1, 0 }] = 0;
    arrayB[Int2{ 2, 0 }] = 0;

    //arrayB[Int2{ 0, 1 }] = 0;
    //arrayB[Int2{ 1, 1 }] = 1;
    //arrayB[Int2{ 2, 1 }] = 0;

    //arrayB[Int2{ 0, 2 }] = 0;
    //arrayB[Int2{ 1, 2 }] = 0;
    //arrayB[Int2{ 2, 2 }] = 1;

    ML_Array arrayC{ Int2{3, 1} };

    arrayA.HostToDevice();
    arrayB.HostToDevice();

    //// Launch the Vector Add CUDA Kernel
    //int threadsPerBlock = 256;
    //int blocksPerGrid = (numElements.Size() + threadsPerBlock - 1) / threadsPerBlock;
    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
    //    threadsPerBlock);
    //vectorAdd << <blocksPerGrid, threadsPerBlock >> > (arrayA.deviceBuffer, arrayB.deviceBuffer, arrayC.deviceBuffer, numElements.Size());
    //err = hipGetLastError();

    //if (err != hipSuccess) {
    //    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
    //        hipGetErrorString(err));
    //    exit(EXIT_FAILURE);
    //}
    
    // Launch the Vector Multiply CUDA Kernel
    Multiply(arrayA, arrayB, arrayC);
    //int threadsPerBlock = 256;
    //int blocksPerGrid = (numElements.Size() + threadsPerBlock - 1) / threadsPerBlock;
    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
    //    threadsPerBlock);
    //vectorMultiply << <blocksPerGrid, threadsPerBlock >> > (arrayA.deviceArray.deviceBuffer, arrayB.deviceArray.deviceBuffer, arrayC.deviceArray.deviceBuffer, numElements);
    //err = hipGetLastError();

    //if (err != hipSuccess) {
    //    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
    //        hipGetErrorString(err));
    //    exit(EXIT_FAILURE);
    //}

    arrayC.DeviceToHost();

    //// Verify that the result vector is correct
    //for (int i = 0; i < numElements.Size(); ++i) {
    //    if (fabs(arrayA.hostBuffer[i] + arrayB.hostBuffer[i] - arrayC.hostBuffer[i]) > 1e-5) {
    //        fprintf(stderr, "Result verification failed at element %d!\n", i);
    //        exit(EXIT_FAILURE);
    //    }
    //}
    // Verify that the result vector is correct
    for (int i = 0; i < numElements.Size(); ++i) {
        if (fabs(arrayA.hostBuffer[i] * arrayB.hostBuffer[i] - arrayC.hostBuffer[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");
}

/**
 * Host main routine
 */
int main(void) {

    Run();

    printf("Done\n");
    return 0;
}
