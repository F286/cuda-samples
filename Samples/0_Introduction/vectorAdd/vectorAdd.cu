#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const float *A, const float *B, float *C,
                          int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements) {
    C[i] = A[i] + B[i] + 0.0f;
  }
}

struct Int2
{
    int x;
    int y;

    int Size()
    {
        return x * y;
    }
};

struct ML_Array
{
    ML_Array(Int2 numElements)
        : numElements(numElements)
        , size(numElements.Size() * sizeof(float))
    {
        // Allocate the host input vector A
        hostArray.resize(numElements.Size());
        hostBuffer = &hostArray[0];

        //h_A = (float*)malloc(size);


        // Verify that allocations succeeded
        if (hostBuffer == NULL) {
            fprintf(stderr, "Failed to allocate host vectors!\n");
            exit(EXIT_FAILURE);
        }

        // Error code to check return values for CUDA calls
        hipError_t err = hipSuccess;

        // Allocate the device input vector A
        err = hipMalloc((void**)&deviceBuffer, size);

        if (err != hipSuccess) {
            fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
                hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    ~ML_Array()
    {
        hipError_t err = hipSuccess;

        // Free device global memory
        err = hipFree(deviceBuffer);

        if (err != hipSuccess) {
            fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
                hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Free host memory
        //free(h_A);
    }

    void InitializeToRandomValues()
    {
        for (int i = 0; i < numElements.Size(); ++i)
        {
            hostBuffer[i] = rand() / (float)RAND_MAX;
        }
    }

    void HostToDevice()
    {
        hipError_t err = hipSuccess;

        // Copy the host input vectors A and B in host memory to the device input
        // vectors in
        // device memory
        printf("Copy input data from the host memory to the CUDA device\n");
        err = hipMemcpy(deviceBuffer, hostBuffer, size, hipMemcpyHostToDevice);

        if (err != hipSuccess) {
            fprintf(stderr,
                "Failed to copy vector A from host to device (error code %s)!\n",
                hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    void DeviceToHost()
    {
        hipError_t err = hipSuccess;

        // Copy the device result vector in device memory to the host result vector
// in host memory.
        printf("Copy output data from the CUDA device to the host memory\n");
        err = hipMemcpy(hostBuffer, deviceBuffer, size, hipMemcpyDeviceToHost);

        if (err != hipSuccess) {
            fprintf(stderr,
                "Failed to copy vector C from device to host (error code %s)!\n",
                hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    // Host
    std::vector<float> hostArray;
    float* hostBuffer;
    // Device
    float* deviceBuffer;

    size_t size;
    Int2 numElements;
};

void Run()
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    Int2 numElements{ 50000, 1 };
    size_t size = numElements.Size() * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements.Size());

    ML_Array arrayA{ numElements };
    arrayA.InitializeToRandomValues();

	ML_Array arrayB{ numElements };
    arrayB.InitializeToRandomValues();

	ML_Array arrayC{ numElements };

    arrayA.HostToDevice();
    arrayB.HostToDevice();

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements.Size() + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
        threadsPerBlock);
    vectorAdd << <blocksPerGrid, threadsPerBlock >> > (arrayA.deviceBuffer, arrayB.deviceBuffer, arrayC.deviceBuffer, numElements.Size());
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    arrayC.DeviceToHost();

    // Verify that the result vector is correct
    for (int i = 0; i < numElements.Size(); ++i) {
        if (fabs(arrayA.hostBuffer[i] + arrayB.hostBuffer[i] - arrayC.hostBuffer[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");
}

/**
 * Host main routine
 */
int main(void) {

    Run();

    printf("Done\n");
    return 0;
}
