#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <assert.h>
#include <memory>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "ML_Array.h"
#include "ML_CheckCudaError.h"
#include "ML_DenseConnection.h"

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
//__global__ void vectorAdd(const float *A, const float *B, float *C,
//                          int numElements) {
//  int i = blockDim.x * blockIdx.x + threadIdx.x;
//
//  if (i < numElements) {
//    C[i] = A[i] + B[i] + 0.0f;
//  }
//}

__global__ void vectorMultiply(const ML_DeviceArray<float> A, const ML_DeviceArray<float> B, ML_DeviceArray<float> C) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < Int2::Size(C.numElements)) 
    {
        float* bufferRootB = &B.deviceBuffer[i * A.numElements.x];

        float total = 0.0f;
        for (int elementIndex = 0; elementIndex < A.numElements.x; elementIndex++)
        {
            total += bufferRootB[elementIndex] * A.deviceBuffer[elementIndex];
        }

        C.deviceBuffer[i] = total;
    }
}

__global__ void vectorDivide(const ML_DeviceArray<float> A, const ML_DeviceArray<float> B, ML_DeviceArray<float> C) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < Int2::Size(C.numElements))
    {
        float* bufferRootB = &B.deviceBuffer[i * A.numElements.x];

        float total = 0.0f;
        for (int elementIndex = 0; elementIndex < A.numElements.x; elementIndex++)
        {
            float b = bufferRootB[elementIndex];
            if (b != 0)
            {
                total += A.deviceBuffer[elementIndex] / b;
            }
        }

        C.deviceBuffer[i] = total;
    }
}

void Multiply(ML_Matrix<float>& input, ML_Matrix<float>& connection, ML_Matrix<float>& output)
{
    input.HostToDevice();
    connection.HostToDevice();

    assert(connection.NumElements().x == input.NumElements().x);
    assert(connection.NumElements().y == output.NumElements().x);

    ML_CheckCudaError checkError;

    Int2 numElements = output.deviceArray.numElements;
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements.Size() + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	vectorMultiply<<<blocksPerGrid, threadsPerBlock>>>(input.deviceArray, connection.deviceArray, output.deviceArray);

    output.DeviceToHost();
}

void Divide(ML_Matrix<float>& input, ML_Matrix<float>& connection, ML_Matrix<float>& output)
{
    input.HostToDevice();
    connection.HostToDevice();

    assert(connection.NumElements().x == input.NumElements().x);
    assert(connection.NumElements().y == output.NumElements().x);

    ML_CheckCudaError checkError;

    Int2 numElements = output.deviceArray.numElements;
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements.Size() + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorDivide << <blocksPerGrid, threadsPerBlock >> > (input.deviceArray, connection.deviceArray, output.deviceArray);

    output.DeviceToHost();
}

void Run()
{
    // Matrix input {10, 100, 100};
    // Matrix connection {{1, 0, 0}, {0, 1, 0}, {0, 0, 1}, {1, 1, 1}};
    // Matrix output = input * connection;

    ML_Matrix<float> array1{ Int2{ 3, 1 } };
    array1[{ 0, 0 }] = 10;
    array1[{ 1, 0 }] = 100;
    array1[{ 2, 0 }] = 1000;

    ML_Matrix<float> array2{ Int2{ 4, 1 } };

    ML_Matrix<float> connection1{ ML_DenseConnection::ConnectionMatrixSize(array1, array2) };

    connection1[{ 0, 0 }] = 1;
    connection1[{ 1, 0 }] = 0;
    connection1[{ 2, 0 }] = 0;

    connection1[{ 0, 1 }] = 0;
    connection1[{ 1, 1 }] = 1;
    connection1[{ 2, 1 }] = 0;

    connection1[{ 0, 2 }] = 0;
    connection1[{ 1, 2 }] = 0;
    connection1[{ 2, 2 }] = 1;

    connection1[{ 0, 3 }] = 1;
    connection1[{ 1, 3 }] = 1;
    connection1[{ 2, 3 }] = 1;

    Multiply(array1, connection1, array2);

    // Verify that the result vector is correct
    assert(array2[0] == 10);
    assert(array2[1] == 100);
    assert(array2[2] == 1000);
    assert(array2[3] == 1110);

    printf("Test PASSED\n");


    ML_Matrix<float> array3{ Int2{ 4, 1 } };
    Divide(array1, connection1, array3);

    assert(array3[0] == 10);
    assert(array3[1] == 100);
    assert(array3[2] == 1000);
    assert(array3[3] == 1110);
}

/**
 * Host main routine
 */
int main(void) {

    Run();

    printf("Done\n");
    return 0;
}
