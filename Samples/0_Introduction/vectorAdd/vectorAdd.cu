#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include "ML_Array.h"

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const float *A, const float *B, float *C,
                          int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements) {
    C[i] = A[i] + B[i] + 0.0f;
  }
}

__global__ void vectorMultiply(const float* A, const float* B, float* C,
    int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] * B[i] + 0.0f;
    }
}

void Run()
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    Int2 numElements{ 2, 1 };
    size_t size = numElements.Size() * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements.Size());

    ML_Array arrayA{ numElements };
    //arrayA.InitializeToRandomValues();
    arrayA[Int2{ 0, 0 }] = 2;
    arrayA[Int2{ 1, 0 }] = 3;

	ML_Array arrayB{ numElements };
    //arrayB.InitializeToRandomValues();
    arrayB[Int2{ 0, 0 }] = 5;
    arrayB[Int2{ 1, 0 }] = 6;

	ML_Array arrayC{ numElements };

    arrayA.HostToDevice();
    arrayB.HostToDevice();

    //// Launch the Vector Add CUDA Kernel
    //int threadsPerBlock = 256;
    //int blocksPerGrid = (numElements.Size() + threadsPerBlock - 1) / threadsPerBlock;
    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
    //    threadsPerBlock);
    //vectorAdd << <blocksPerGrid, threadsPerBlock >> > (arrayA.deviceBuffer, arrayB.deviceBuffer, arrayC.deviceBuffer, numElements.Size());
    //err = hipGetLastError();

    //if (err != hipSuccess) {
    //    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
    //        hipGetErrorString(err));
    //    exit(EXIT_FAILURE);
    //}
    
    // Launch the Vector Multiply CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements.Size() + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
        threadsPerBlock);
    vectorMultiply << <blocksPerGrid, threadsPerBlock >> > (arrayA.deviceBuffer, arrayB.deviceBuffer, arrayC.deviceBuffer, numElements.Size());
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    arrayC.DeviceToHost();

    //// Verify that the result vector is correct
    //for (int i = 0; i < numElements.Size(); ++i) {
    //    if (fabs(arrayA.hostBuffer[i] + arrayB.hostBuffer[i] - arrayC.hostBuffer[i]) > 1e-5) {
    //        fprintf(stderr, "Result verification failed at element %d!\n", i);
    //        exit(EXIT_FAILURE);
    //    }
    //}
    // Verify that the result vector is correct
    for (int i = 0; i < numElements.Size(); ++i) {
        if (fabs(arrayA.hostBuffer[i] * arrayB.hostBuffer[i] - arrayC.hostBuffer[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");
}

/**
 * Host main routine
 */
int main(void) {

    Run();

    printf("Done\n");
    return 0;
}
